#include "hip/hip_runtime.h"
#include "include/euclidean_cluster.h"
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/fill.h>

GpuEuclideanCluster2::GpuEuclideanCluster2()
{
	x_ = y_ = z_ = NULL;

	point_num_ = 0;
	threshold_ = 0;
	cluster_name_ = NULL;
	cluster_name_host_ = NULL;
	min_cluster_pts_ = 0;
	max_cluster_pts_ = INT_MAX;
	cluster_num_ = 0;
}

void GpuEuclideanCluster2::setThreshold(double threshold)
{
	threshold_ = threshold;
}

void GpuEuclideanCluster2::setMinClusterPts(int min_cluster_pts)
{
	min_cluster_pts_ = min_cluster_pts;
}

void GpuEuclideanCluster2::setMaxClusterPts(int max_cluster_pts)
{
	max_cluster_pts_ = max_cluster_pts;
}

__global__ void convertFormat(pcl::PointXYZ *input, float *out_x, float *out_y, float *out_z, int point_num)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = idx; i < point_num; i += blockDim.x * gridDim.x) {
		pcl::PointXYZ tmp_input = input[i];
		out_x[i] = tmp_input.x;
		out_y[i] = tmp_input.y;
		//out_z[i] = tmp_input.z;
		// Convert to 2d cloud
		out_z[i] = 0;
	}
}

void GpuEuclideanCluster2::exclusiveScan(int *input, int ele_num, int *sum)
{
	thrust::device_ptr<int> dev_ptr(input);

	thrust::exclusive_scan(dev_ptr, dev_ptr + ele_num, dev_ptr);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	*sum = *(dev_ptr + ele_num - 1);
}

void GpuEuclideanCluster2::setInputPoints(pcl::PointCloud<pcl::PointXYZ>::Ptr input)
{
	if (input->size() > 0) {
		point_num_ = input->size();
		checkCudaErrors(hipMalloc(&x_, sizeof(float) * point_num_));
		checkCudaErrors(hipMalloc(&y_, sizeof(float) * point_num_));
		checkCudaErrors(hipMalloc(&z_, sizeof(float) * point_num_));

		pcl::PointXYZ *dev_tmp_input;

		checkCudaErrors(hipMalloc(&dev_tmp_input, sizeof(pcl::PointXYZ) * point_num_));
		checkCudaErrors(hipMemcpy(dev_tmp_input, input->points.data(), sizeof(pcl::PointXYZ) * point_num_, hipMemcpyHostToDevice));

		int block_x = (point_num_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : point_num_;
		int grid_x = (point_num_ - 1) / block_x + 1;

		convertFormat<<<grid_x, block_x>>>(dev_tmp_input, x_, y_, z_, point_num_);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipFree(dev_tmp_input));


		checkCudaErrors(hipMalloc(&cluster_name_, point_num_ * sizeof(int)));
		cluster_name_host_ = (int*)malloc(point_num_ * sizeof(int));
	}
}

__global__ void initClusterNames(int *cluster_names, int point_num)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < point_num; i += blockDim.x * gridDim.x) {
		cluster_names[i] = i;
	}
}

void GpuEuclideanCluster2::initClusters()
{
	int block_x, grid_x;

	block_x = (point_num_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : point_num_;
	grid_x = (point_num_ - 1) / block_x + 1;

	initClusterNames<<<grid_x, block_x>>>(cluster_name_, point_num_);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

std::vector<GpuEuclideanCluster2::GClusterIndex> GpuEuclideanCluster2::getOutput()
{
	std::vector<GpuEuclideanCluster2::GClusterIndex> output(cluster_num_);

	if (cluster_num_ == 0)
		return output;

	for (int i = 0; i < cluster_num_; i++) {
		output[i].index_value = i;
	}

	for (int i = 0; i < point_num_; i++) {
		GClusterIndex &cluster = output[cluster_name_host_[i]];

		cluster.points_in_cluster.push_back(i);
	}

	int point_num_test = 0;

	for (unsigned int i = 0; i < output.size();) {
		int number_of_pts = output[i].points_in_cluster.size();

		point_num_test += number_of_pts;

		if (number_of_pts < min_cluster_pts_ || number_of_pts > max_cluster_pts_)
			output.erase(output.begin() + i);
		else
			i++;
	}

	return output;
}

/* Re-indexing the cluster name array to make all cluster indexes starts from 0 */
__global__ void renameClusters(int *cluster_name, int *cluster_location, int point_num)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < point_num; i += blockDim.x * gridDim.x) {
		int old_name = cluster_name[i];

		cluster_name[i] = cluster_location[old_name];
	}
}

void GpuEuclideanCluster2::renamingClusters(int *cluster_names, int *cluster_location, int point_num) {
	int block_x = (point_num > BLOCK_SIZE_X) ? BLOCK_SIZE_X : point_num;
	int grid_x = (point_num - 1) / block_x + 1;

	renameClusters<<<grid_x, block_x>>>(cluster_names, cluster_location, point_num);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}


GpuEuclideanCluster2::~GpuEuclideanCluster2()
{
	if (x_ != NULL) {
		checkCudaErrors(hipFree(x_));
	}

	if (y_ != NULL) {
		checkCudaErrors(hipFree(y_));
	}

	if (z_ != NULL) {
		checkCudaErrors(hipFree(z_));
	}

	if (cluster_name_ != NULL) {
		checkCudaErrors(hipFree(cluster_name_));
	}

	if (cluster_name_host_ != NULL) {
		free(cluster_name_host_);
	}
}

