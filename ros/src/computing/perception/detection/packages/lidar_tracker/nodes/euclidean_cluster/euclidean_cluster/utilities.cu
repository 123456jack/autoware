#include "include/utilities.h"
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/fill.h>


void GUtilities::exclusiveScan(int *input, int ele_num)
{
	exclusiveScan<int>(input, ele_num);
}

void GUtilities::exclusiveScan(int *input, int ele_num, int *sum)
{
	exclusiveScan<int>(input, ele_num, sum);
}

void GUtilities::exclusiveScan(long long int *input, int ele_num, long long int *sum)
{
	exclusiveScan<long long int>(input, ele_num, sum);
}

void GUtilities::exclusiveScan(unsigned long long int *input, int ele_num, unsigned long long int *sum)
{
	exclusiveScan<unsigned long long int>(input, ele_num, sum);
}

template <typename T>
void GUtilities::exclusiveScan(T *input, int ele_num, T *sum)
{
	thrust::device_ptr<T> dev_ptr(input);

	thrust::exclusive_scan(dev_ptr, dev_ptr + ele_num, dev_ptr);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	*sum = *(dev_ptr + ele_num - 1);
}

template <typename T>
void GUtilities::exclusiveScan(T *input, int ele_num)
{
	thrust::device_ptr<T> dev_ptr(input);

	thrust::exclusive_scan(dev_ptr, dev_ptr + ele_num, dev_ptr);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

