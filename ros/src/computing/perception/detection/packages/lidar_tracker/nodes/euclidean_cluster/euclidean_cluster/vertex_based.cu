#include "hip/hip_runtime.h"
#include "include/euclidean_cluster.h"
#include <hip/hip_runtime.h>

#define TEST_VERTEX_ 1

extern __shared__ float local_buff[];

__global__ void frontierInitialize(int *frontier_array, int point_num)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < point_num; i += blockDim.x * gridDim.x) {
		frontier_array[i] = 1;
	}
}

__global__ void countAdjacentList(float *x, float *y, float *z, int point_num, float threshold, long long int *adjacent_count)
{
	float *local_x = local_buff;
	float *local_y = local_x + blockDim.x;
	float *local_z = local_y + blockDim.x;
	int pid;
	int last_point = (point_num / blockDim.x) * blockDim.x;	// Exclude the last block
	float dist;

	for (pid = threadIdx.x + blockIdx.x * blockDim.x; pid < last_point; pid += blockDim.x * gridDim.x) {
		float tmp_x = x[pid];
		float tmp_y = y[pid];
		float tmp_z = z[pid];
		int count = 0;

		int block_id;

		for (block_id = 0; block_id + blockDim.x < point_num; block_id += blockDim.x) {
			local_x[threadIdx.x] = x[block_id + threadIdx.x];
			local_y[threadIdx.x] = y[block_id + threadIdx.x];
			local_z[threadIdx.x] = z[block_id + threadIdx.x];
			__syncthreads();

			for (int i = 0; i < blockDim.x; i++) {
				dist = norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]);
				count += (i + block_id != pid && dist < threshold) ? 1 : 0;
			}
			__syncthreads();
		}

		__syncthreads();

		// Compare with last block
		if (threadIdx.x < point_num - block_id) {
			local_x[threadIdx.x] = x[block_id + threadIdx.x];
			local_y[threadIdx.x] = y[block_id + threadIdx.x];
			local_z[threadIdx.x] = z[block_id + threadIdx.x];
		}
		__syncthreads();

		for (int i = 0; i < point_num - block_id; i++) {
			dist = norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]);
			count += (i + block_id != pid && dist < threshold) ? 1 : 0;
		}

		adjacent_count[pid] = count;
		__syncthreads();
	}
	__syncthreads();


	// Handle last block
	if (pid >= last_point) {
		int count = 0;
		float tmp_x, tmp_y, tmp_z;

		if (pid < point_num) {
			tmp_x = x[pid];
			tmp_y = y[pid];
			tmp_z = z[pid];
		}

		__syncthreads();

		int block_id;

		for (block_id = 0; block_id + blockDim.x < point_num; block_id += blockDim.x) {
			local_x[threadIdx.x] = x[block_id + threadIdx.x];
			local_y[threadIdx.x] = y[block_id + threadIdx.x];
			local_z[threadIdx.x] = z[block_id + threadIdx.x];
			__syncthreads();

			if (pid < point_num) {
				for (int i = 0; i < blockDim.x; i++) {
					dist = norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]);
					count += (i + block_id != pid && dist < threshold) ? 1 : 0;
				}
			}
			__syncthreads();
		}
		__syncthreads();

		if (pid < point_num) {
			local_x[threadIdx.x] = x[pid];
			local_y[threadIdx.x] = y[pid];
			local_z[threadIdx.x] = z[pid];
			__syncthreads();

			for (int i = 0; i < point_num - block_id; i++) {
				dist = norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]);
				count += (i + block_id != pid && dist < threshold) ? 1 : 0;
			}
			__syncthreads();

			adjacent_count[pid] = count;
		}
	}
}

__global__ void buildAdjacentList(float *x, float *y, float *z, int point_num, float threshold, long long int *adjacent_count, int *adjacent_list)
{
	float *local_x = local_buff;
	float *local_y = local_x + blockDim.x;
	float *local_z = local_y + blockDim.x;
	int pid;
	int last_point = (point_num / blockDim.x) * blockDim.x;

	for (pid = threadIdx.x + blockIdx.x * blockDim.x; pid < last_point; pid += blockDim.x * gridDim.x) {
		long long int writing_location = adjacent_count[pid];
		float tmp_x = x[pid];
		float tmp_y = y[pid];
		float tmp_z = z[pid];

		int block_id;

		for (block_id = 0; block_id + blockDim.x < point_num; block_id += blockDim.x) {
			local_x[threadIdx.x] = x[block_id + threadIdx.x];
			local_y[threadIdx.x] = y[block_id + threadIdx.x];
			local_z[threadIdx.x] = z[block_id + threadIdx.x];
			__syncthreads();

			for (int i = 0; i < blockDim.x; i++) {
				if (i + block_id != pid && norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]) < threshold) {
					adjacent_list[writing_location++] = i + block_id;
				}
			}
			__syncthreads();
		}
		__syncthreads();


		if (threadIdx.x < point_num - block_id) {
			local_x[threadIdx.x] = x[block_id + threadIdx.x];
			local_y[threadIdx.x] = y[block_id + threadIdx.x];
			local_z[threadIdx.x] = z[block_id + threadIdx.x];
		}
		__syncthreads();

		for (int i = 0; i < point_num - block_id; i++) {
			if (i + block_id != pid && norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]) < threshold) {
				adjacent_list[writing_location++] = i + block_id;
			}
		}
		__syncthreads();

	}

	if (pid >= last_point) {
		float tmp_x, tmp_y, tmp_z;
		int writing_location;

		if (pid < point_num) {
			tmp_x = x[pid];
			tmp_y = y[pid];
			tmp_z = z[pid];
			writing_location = adjacent_count[pid];
		}

		int block_id;

		for (block_id = 0; block_id + blockDim.x < point_num; block_id += blockDim.x) {
			local_x[threadIdx.x] = x[block_id + threadIdx.x];
			local_y[threadIdx.x] = y[block_id + threadIdx.x];
			local_z[threadIdx.x] = z[block_id + threadIdx.x];
			__syncthreads();

			if (pid < point_num) {
				for (int i = 0; i < blockDim.x; i++) {
					if (i + block_id != pid && norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]) < threshold) {
						adjacent_list[writing_location++] = i + block_id;
					}
				}
			}
			__syncthreads();
		}
		__syncthreads();

		if (pid < point_num) {
			local_x[threadIdx.x] = x[pid];
			local_y[threadIdx.x] = y[pid];
			local_z[threadIdx.x] = z[pid];
			__syncthreads();

			for (int i = 0; i < point_num - block_id; i++) {
				if (i + block_id != pid && norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]) < threshold) {
					adjacent_list[writing_location++] = i + block_id;
				}
			}
		}
	}
}

__global__ void clustering(long long int *adjacent_list_loc, int *adjacent_list, int point_num, int *cluster_name, int *frontier_array1, int *frontier_array2, bool *changed)
{
	__shared__ bool schanged;

	if (threadIdx.x == 0)
		schanged = false;
	__syncthreads();

	for (int pid = threadIdx.x + blockIdx.x * blockDim.x; pid < point_num; pid += blockDim.x * gridDim.x) {
		if (frontier_array1[pid] == 1) {
			frontier_array1[pid] = 0;
			int cname = cluster_name[pid];
			bool c = false;
			long long int start = adjacent_list_loc[pid];
			long long int end = adjacent_list_loc[pid + 1];

			// Iterate through neighbors' ids
			for (long long int i = start; i < end; i++) {
				int nid = adjacent_list[i];
				int nname = cluster_name[nid];
				if (cname < nname) {
					atomicMin(cluster_name + nid, cname);
					frontier_array2[nid] = 1;
					schanged = true;
					//*changed = true;
				} else if (cname > nname) {
					cname = nname;
					c = true;
				}
			}

			if (c) {
				atomicMin(cluster_name + pid, cname);
				frontier_array2[pid] = 1;
				schanged = true;
				//*changed = true;
			}
		}
	}
	__syncthreads();

	if (threadIdx.x == 0 && schanged)
		*changed = true;
}

/* Iterate through the list of remaining clusters and mark the corresponding
 * location on cluster location array by 1
 */
__global__ void clusterMark2(int *cluster_list, int *cluster_location, int cluster_num)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = idx; i < cluster_num; i += blockDim.x * gridDim.x) {
		cluster_location[cluster_list[i]] = 1;
	}
}

void GpuEuclideanCluster2::extractClusters2()
{
	initClusters();

	int block_x = (point_num_ < block_size_x_) ? point_num_ : block_size_x_;
	int grid_x = (point_num_ - 1) / block_x + 1;

	long long int *adjacent_count;
	int *adjacent_list;

	checkCudaErrors(hipMalloc(&adjacent_count, sizeof(long long int) * (point_num_ + 1)));

#ifdef TEST_VERTEX_
	struct timeval start, end;

	gettimeofday(&start, NULL);
#endif

	countAdjacentList<<<grid_x, block_x, sizeof(float) * block_size_x_ * 3>>>(x_, y_, z_, point_num_, threshold_, adjacent_count);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

#ifdef TEST_VERTEX_
	gettimeofday(&end, NULL);

	std::cout << "Count ADJ = " << timeDiff(start, end) << std::endl;
#endif

	long long int adjacent_list_size;

	exclusiveScan(adjacent_count, point_num_ + 1, &adjacent_list_size);

	if (adjacent_list_size == 0) {
		checkCudaErrors(hipFree(adjacent_count));
		cluster_num_ = 0;
		return;
	}
	checkCudaErrors(hipMalloc(&adjacent_list, sizeof(int) * adjacent_list_size));


#ifdef TEST_VERTEX_
	gettimeofday(&start, NULL);
#endif

	buildAdjacentList<<<grid_x, block_x, sizeof(float) * block_size_x_ * 3>>>(x_, y_, z_, point_num_, threshold_, adjacent_count, adjacent_list);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

#ifdef TEST_VERTEX_
	gettimeofday(&end, NULL);

	std::cout << "Build ADJ = " << timeDiff(start, end) << std::endl;
#endif

#define HOST_ALLOC_

	bool *changed;

#ifndef HOST_ALLOC_
	bool hchanged;
	checkCudaErrors(hipMalloc(&changed, sizeof(bool)));
#else
	checkCudaErrors(hipHostMalloc(&changed, sizeof(bool)));
#endif

	int *frontier_array1, *frontier_array2;

	checkCudaErrors(hipMalloc(&frontier_array1, sizeof(int) * point_num_));
	checkCudaErrors(hipMalloc(&frontier_array2, sizeof(int) * point_num_));

	frontierInitialize<<<grid_x, block_x>>>(frontier_array1, point_num_);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemset(frontier_array2, 0, sizeof(int) * point_num_));
	checkCudaErrors(hipDeviceSynchronize());

#ifdef TEST_VERTEX_
	gettimeofday(&start, NULL);
#endif

	int itr = 0;

	do {
#ifndef HOST_ALLOC_
		hchanged = false;
		checkCudaErrors(hipMemcpy(changed, &hchanged, sizeof(bool), hipMemcpyHostToDevice));
#else
		*changed = false;
#endif

		clustering<<<grid_x, block_x>>>(adjacent_count, adjacent_list, point_num_, cluster_name_, frontier_array1, frontier_array2, changed);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		int *tmp;

		tmp = frontier_array1;
		frontier_array1 = frontier_array2;
		frontier_array2 = tmp;
#ifndef HOST_ALLOC_
		checkCudaErrors(hipMemcpy(&hchanged, changed, sizeof(bool), hipMemcpyDeviceToHost));
	} while (hchanged);
#else
		itr++;
	} while (*changed);
#endif

#ifdef TEST_VERTEX_
	gettimeofday(&end, NULL);

	std::cout << "Iteration = " << timeDiff(start, end) << std::endl;
#endif

	std::cout << "Iteration num = " << itr << std::endl;

	// renaming clusters
	int *cluster_location;

	checkCudaErrors(hipMalloc(&cluster_location, sizeof(int) * (point_num_ + 1)));
	checkCudaErrors(hipMemset(cluster_location, 0, sizeof(int) * (point_num_ + 1)));

	clusterMark2<<<grid_x, block_x>>>(cluster_name_, cluster_location, point_num_);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	exclusiveScan(cluster_location, point_num_ + 1, &cluster_num_);

	renamingClusters(cluster_name_, cluster_location, point_num_);

	checkCudaErrors(hipMemcpy(cluster_name_host_, cluster_name_, sizeof(int) * point_num_, hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(adjacent_count));
	checkCudaErrors(hipFree(adjacent_list));
	checkCudaErrors(hipFree(frontier_array1));
	checkCudaErrors(hipFree(frontier_array2));
#ifndef HOST_ALLOC_
	checkCudaErrors(hipFree(changed));
#else
	checkCudaErrors(hipHostFree(changed));
#endif
	checkCudaErrors(hipFree(cluster_location));
}

