#include "hip/hip_runtime.h"
#include "include/euclidean_cluster.h"
#include <hip/hip_runtime.h>


__global__ void frontierInitialize(int *frontier_array, int point_num)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < point_num; i += blockDim.x * gridDim.x) {
		frontier_array[i] = 1;
	}
}

__global__ void countAdjacentList(float *x, float *y, float *z, int point_num, float threshold, int *adjacent_count)
{
	__shared__ float local_x[BLOCK_SIZE_X];
	__shared__ float local_y[BLOCK_SIZE_X];
	__shared__ float local_z[BLOCK_SIZE_X];
	int pid;
	int last_point = (point_num / blockDim.x) * blockDim.x;	// Exclude the last block
	float dist;

	for (pid = threadIdx.x + blockIdx.x * blockDim.x; pid < last_point; pid += blockDim.x * gridDim.x) {
		float tmp_x = x[pid];
		float tmp_y = y[pid];
		float tmp_z = z[pid];
		int count = 0;

		int block_id;

		for (block_id = 0; block_id + blockDim.x < point_num; block_id += blockDim.x) {
			local_x[threadIdx.x] = x[block_id + threadIdx.x];
			local_y[threadIdx.x] = y[block_id + threadIdx.x];
			local_z[threadIdx.x] = z[block_id + threadIdx.x];
			__syncthreads();

			for (int i = 0; i < blockDim.x; i++) {
				dist = norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]);
				count += (i + block_id != pid && dist < threshold) ? 1 : 0;
			}
			__syncthreads();
		}

		__syncthreads();

		// Compare with last block
		if (threadIdx.x < point_num - block_id) {
			local_x[threadIdx.x] = x[block_id + threadIdx.x];
			local_y[threadIdx.x] = y[block_id + threadIdx.x];
			local_z[threadIdx.x] = z[block_id + threadIdx.x];
		}
		__syncthreads();

		for (int i = 0; i < point_num - block_id; i++) {
			dist = norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]);
			count += (i + block_id != pid && dist < threshold) ? 1 : 0;
		}

		adjacent_count[pid] = count;
		__syncthreads();
	}
	__syncthreads();


	// Handle last block
	if (pid >= last_point) {
		int count = 0;
		float tmp_x, tmp_y, tmp_z;

		if (pid < point_num) {
			tmp_x = x[pid];
			tmp_y = y[pid];
			tmp_z = z[pid];
		}

		__syncthreads();

		int block_id;

		for (block_id = 0; block_id + blockDim.x < point_num; block_id += blockDim.x) {
			local_x[threadIdx.x] = x[block_id + threadIdx.x];
			local_y[threadIdx.x] = y[block_id + threadIdx.x];
			local_z[threadIdx.x] = z[block_id + threadIdx.x];
			__syncthreads();

			if (pid < point_num) {
				for (int i = 0; i < blockDim.x; i++) {
					dist = norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]);
					count += (i + block_id != pid && dist < threshold) ? 1 : 0;
				}
			}
			__syncthreads();
		}
		__syncthreads();

		if (pid < point_num) {
			local_x[threadIdx.x] = x[pid];
			local_y[threadIdx.x] = y[pid];
			local_z[threadIdx.x] = z[pid];
			__syncthreads();

			for (int i = 0; i < point_num - block_id; i++) {
				dist = norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]);
				count += (i + block_id != pid && dist < threshold) ? 1 : 0;
			}
			__syncthreads();

			adjacent_count[pid] = count;
		}
	}
}

__global__ void buildAdjacentList(float *x, float *y, float *z, int point_num, float threshold, int *adjacent_count, int *adjacent_list)
{
	__shared__ float local_x[BLOCK_SIZE_X];
	__shared__ float local_y[BLOCK_SIZE_X];
	__shared__ float local_z[BLOCK_SIZE_X];
	int pid;
	int last_point = (point_num / blockDim.x) * blockDim.x;

	for (pid = threadIdx.x + blockIdx.x * blockDim.x; pid < last_point; pid += blockDim.x * gridDim.x) {
		int writing_location = adjacent_count[pid];
		float tmp_x = x[pid];
		float tmp_y = y[pid];
		float tmp_z = z[pid];

		int block_id;

		for (block_id = 0; block_id + blockDim.x < point_num; block_id += blockDim.x) {
			local_x[threadIdx.x] = x[block_id + threadIdx.x];
			local_y[threadIdx.x] = y[block_id + threadIdx.x];
			local_z[threadIdx.x] = z[block_id + threadIdx.x];
			__syncthreads();

			for (int i = 0; i < blockDim.x; i++) {
				if (i + block_id != pid && norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]) < threshold) {
					adjacent_list[writing_location++] = i + block_id;
				}
			}
			__syncthreads();
		}
		__syncthreads();


		if (threadIdx.x < point_num - block_id) {
			local_x[threadIdx.x] = x[block_id + threadIdx.x];
			local_y[threadIdx.x] = y[block_id + threadIdx.x];
			local_z[threadIdx.x] = z[block_id + threadIdx.x];
		}
		__syncthreads();

		for (int i = 0; i < point_num - block_id; i++) {
			if (i + block_id != pid && norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]) < threshold) {
				adjacent_list[writing_location++] = i + block_id;
			}
		}
		__syncthreads();

	}

	if (pid >= last_point) {
		float tmp_x, tmp_y, tmp_z;
		int writing_location;

		if (pid < point_num) {
			tmp_x = x[pid];
			tmp_y = y[pid];
			tmp_z = z[pid];
			writing_location = adjacent_count[pid];
		}

		int block_id;

		for (block_id = 0; block_id + blockDim.x < point_num; block_id += blockDim.x) {
			local_x[threadIdx.x] = x[block_id + threadIdx.x];
			local_y[threadIdx.x] = y[block_id + threadIdx.x];
			local_z[threadIdx.x] = z[block_id + threadIdx.x];
			__syncthreads();

			if (pid < point_num) {
				for (int i = 0; i < blockDim.x; i++) {
					if (i + block_id != pid && norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]) < threshold) {
						adjacent_list[writing_location++] = i + block_id;
					}
				}
			}
			__syncthreads();
		}
		__syncthreads();

		if (pid < point_num) {
			local_x[threadIdx.x] = x[pid];
			local_y[threadIdx.x] = y[pid];
			local_z[threadIdx.x] = z[pid];
			__syncthreads();

			for (int i = 0; i < point_num - block_id; i++) {
				if (i + block_id != pid && norm3df(tmp_x - local_x[i], tmp_y - local_y[i], tmp_z - local_z[i]) < threshold) {
					adjacent_list[writing_location++] = i + block_id;
				}
			}
		}
	}
}

__global__ void clustering(int *adjacent_list_loc, int *adjacent_list, int point_num, int *cluster_name, int *frontier_array1, int *frontier_array2, bool *changed)
{
	__shared__ bool schanged;

	if (threadIdx.x == 0)
		schanged = false;
	__syncthreads();

	for (int pid = threadIdx.x + blockIdx.x * blockDim.x; pid < point_num; pid += blockDim.x * gridDim.x) {
		if (frontier_array1[pid] == 1) {
			frontier_array1[pid] = 0;
			int cname = cluster_name[pid];
			bool c = false;
			int start = adjacent_list_loc[pid];
			int end = adjacent_list_loc[pid + 1];

			// Iterate through neighbors' ids
			for (int i = start; i < end; i++) {
				int nid = adjacent_list[i];
				int nname = cluster_name[nid];
				if (cname < nname) {
					atomicMin(cluster_name + nid, cname);
					frontier_array2[nid] = 1;
					schanged = true;
					//*changed = true;
				} else if (cname > nname) {
					cname = nname;
					c = true;
				}
			}

			if (c) {
				atomicMin(cluster_name + pid, cname);
				frontier_array2[pid] = 1;
				schanged = true;
				//*changed = true;
			}
		}
	}
	__syncthreads();

	if (threadIdx.x == 0 && schanged)
		*changed = true;
}

/* Iterate through the list of remaining clusters and mark the corresponding
 * location on cluster location array by 1
 */
__global__ void clusterMark2(int *cluster_list, int *cluster_location, int cluster_num)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = idx; i < cluster_num; i += blockDim.x * gridDim.x) {
		cluster_location[cluster_list[i]] = 1;
	}
}

void GpuEuclideanCluster2::extractClusters2()
{
	initClusters();

	int block_x = (point_num_ < BLOCK_SIZE_X) ? point_num_ : BLOCK_SIZE_X;
	int grid_x = (point_num_ - 1) / block_x + 1;

	int *adjacent_count, *adjacent_list;

	checkCudaErrors(hipMalloc(&adjacent_count, sizeof(int) * (point_num_ + 1)));

	struct timeval start, end;

	gettimeofday(&start, NULL);
	countAdjacentList<<<grid_x, block_x>>>(x_, y_, z_, point_num_, threshold_, adjacent_count);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	gettimeofday(&end, NULL);

	std::cout << "Count ADJ = " << timeDiff(start, end) << std::endl;
	int adjacent_list_size;

	exclusiveScan(adjacent_count, point_num_ + 1, &adjacent_list_size);

	if (adjacent_list_size == 0) {
		checkCudaErrors(hipFree(adjacent_count));
		cluster_num_ = 0;
		return;
	}

	checkCudaErrors(hipMalloc(&adjacent_list, sizeof(int) * adjacent_list_size));

	gettimeofday(&start, NULL);
	buildAdjacentList<<<grid_x, block_x>>>(x_, y_, z_, point_num_, threshold_, adjacent_count, adjacent_list);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	gettimeofday(&end, NULL);

	std::cout << "Build ADJ = " << timeDiff(start, end) << std::endl;

#define HOST_ALLOC_

	bool *changed;

#ifndef HOST_ALLOC_
	bool hchanged;
	checkCudaErrors(hipMalloc(&changed, sizeof(bool)));
#else
	checkCudaErrors(hipHostMalloc(&changed, sizeof(bool)));
#endif

	int *frontier_array1, *frontier_array2;

	checkCudaErrors(hipMalloc(&frontier_array1, sizeof(int) * point_num_));
	checkCudaErrors(hipMalloc(&frontier_array2, sizeof(int) * point_num_));

	frontierInitialize<<<grid_x, block_x>>>(frontier_array1, point_num_);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemset(frontier_array2, 0, sizeof(int) * point_num_));
	checkCudaErrors(hipDeviceSynchronize());

	gettimeofday(&start, NULL);
	do {
#ifndef HOST_ALLOC_
		hchanged = false;
		checkCudaErrors(hipMemcpy(changed, &hchanged, sizeof(bool), hipMemcpyHostToDevice));
#else
		*changed = false;
#endif

		clustering<<<grid_x, block_x>>>(adjacent_count, adjacent_list, point_num_, cluster_name_, frontier_array1, frontier_array2, changed);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		int *tmp;

		tmp = frontier_array1;
		frontier_array1 = frontier_array2;
		frontier_array2 = tmp;
#ifndef HOST_ALLOC_
		checkCudaErrors(hipMemcpy(&hchanged, changed, sizeof(bool), hipMemcpyDeviceToHost));
	} while (hchanged);
#else
} while (*changed);
#endif

gettimeofday(&end, NULL);

std::cout << "Iteration = " << timeDiff(start, end) << std::endl;

	// renaming clusters
	int *cluster_location;

	checkCudaErrors(hipMalloc(&cluster_location, sizeof(int) * (point_num_ + 1)));
	checkCudaErrors(hipMemset(cluster_location, 0, sizeof(int) * (point_num_ + 1)));

	clusterMark2<<<grid_x, block_x>>>(cluster_name_, cluster_location, point_num_);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	exclusiveScan(cluster_location, point_num_ + 1, &cluster_num_);

	renamingClusters(cluster_name_, cluster_location, point_num_);

	checkCudaErrors(hipMemcpy(cluster_name_host_, cluster_name_, sizeof(int) * point_num_, hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(adjacent_count));
	checkCudaErrors(hipFree(adjacent_list));
	checkCudaErrors(hipFree(frontier_array1));
	checkCudaErrors(hipFree(frontier_array2));
#ifndef HOST_ALLOC_
	checkCudaErrors(hipFree(changed));
#else
	checkCudaErrors(hipHostFree(changed));
#endif
	checkCudaErrors(hipFree(cluster_location));
}

