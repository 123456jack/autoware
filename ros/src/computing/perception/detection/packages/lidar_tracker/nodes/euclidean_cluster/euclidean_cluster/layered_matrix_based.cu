#include "hip/hip_runtime.h"
#include "include/euclidean_cluster.h"
#include <hip/hip_runtime.h>

#define NON_ATOMIC_ 1

/* Connected component labeling points at GPU block thread level.
 * Input list of points is divided into multiple smaller groups.
 * Each group of point is assigned to a block of GPU thread.
 * Each thread in a block handles one point in the group. It iterates over
 * points in the group and compare the distance between the current point A
 * and the point B it has to handle.
 *
 * If the distance between A and B is less than the threshold, then those
 * two points belong to a same connected component and the cluster_changed
 * is marked by 1.
 *
 * A synchronization is called to make sure all thread in the block finish A
 * before moving to the update phase.
 * After finishing checking cluster_changed, threads update the cluster
 * index of all points. If a thread has cluster_changed is 1, then the corresponding
 * cluster of the point it is handling is changed to the cluster of B. Otherwise
 * the original cluster of A remains unchanged.
 *
 * Another synchronization is called before all threads in the block move to
 * other points after done checking A.
 *
 * After this kernel finishes, all points in each block are labeled.
 */


/* Arrays to remember:
 * cluster_name: the index of the cluster that each point belong to
 * 				i.e. point at index i belong to cluster cluster_name[i]
 * cluster_list: the list of remaining clusters
 * cluster_location: location of the remaining clusters in the cluster list
 * 					i.e. cluster A locate in index cluster_location[A] in the
 * 					cluster_list
 * matrix: the adjacency matrix of the cluster list, each cluster is a vertex.
 * 			This matrix is rebuilt whenever some clusters are merged together
 */

/* The adjacency matrix is divided into non-overlap sub-matrices each
 * contains 1024 consecutive columns and 1024 consecutive rows. Empty
 * sub-matrices are ignored.
 *
 * A sample matrix is created to store the status of each sub-matrix.
 * A cell of it is 0 if the corresponding sub-matrix is zero and 1 otherwise.
 *
 * The sub_mat_location array is the result of a prefix sum on the sample
 * matrix. It is used to access non-zero sub-matrices.
 *
 * Finally, the actual non-zero sub-matrices are stored in a matrix,
 * which is an int array.
 */

__global__ void blockClusteringM4(float *x, float *y, float *z, int point_num, int *cluster_name, float threshold)
{
	int block_start = blockIdx.x * blockDim.x;
	int block_end = (block_start + blockDim.x > point_num) ? point_num : block_start + blockDim.x;
	__shared__ float local_x[BLOCK_SIZE_X];
	__shared__ float local_y[BLOCK_SIZE_X];
	__shared__ float local_z[BLOCK_SIZE_X];
	/* Each thread is in charge of one point in the block.*/
	int pid = threadIdx.x + block_start;
	/* Local cluster to record the change in the name of the cluster each point belong to */
	__shared__ int local_cluster_idx[BLOCK_SIZE_X];
	/* Cluster changed to check if a cluster name has changed after each comparison */
	__shared__ int cluster_changed[BLOCK_SIZE_X];

	if (pid < block_end) {
		local_cluster_idx[threadIdx.x] = threadIdx.x;
		local_x[threadIdx.x] = x[pid];
		local_y[threadIdx.x] = y[pid];
		local_z[threadIdx.x] = z[pid];
		__syncthreads();

		float cx = local_x[threadIdx.x];
		float cy = local_y[threadIdx.x];
		float cz = local_z[threadIdx.x];

		/* Iterate through all points in the block and check if the point at row index
		 * and at column index belong to the same cluster.
		 * If so, then name of the cluster of the row point is changed into the name
		 * of the cluster of column point.
		 * */
		for (int rid = 0; rid < block_end - block_start; rid++) {
			float distance = norm3df(cx - local_x[rid], cy - local_y[rid], cz - local_z[rid]);
			int row_cluster = local_cluster_idx[rid];
			int col_cluster = local_cluster_idx[threadIdx.x];

			cluster_changed[threadIdx.x] = 0;
			__syncthreads();

			if (threadIdx.x > rid && distance < threshold) {
				cluster_changed[col_cluster] = 1;
			}
			__syncthreads();

			local_cluster_idx[threadIdx.x] = (cluster_changed[col_cluster] == 1) ? row_cluster : col_cluster;
			__syncthreads();
		}
		__syncthreads();

		int new_cluster = cluster_name[block_start + local_cluster_idx[threadIdx.x]];
		__syncthreads();

		cluster_name[pid] = new_cluster;
	}
}

__global__ void nonZeroSubMatrixCountM4(float *x, float *y, float *z, int point_num,
										int *cluster_name, int *cluster_location,
										int *sample_matrix, int sample_size,
										int cluster_num, float threshold)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int cpid = idx; cpid < point_num; cpid += stride) {
		float tmp_x = x[cpid];
		float tmp_y = y[cpid];
		float tmp_z = z[cpid];
		int col_cluster = cluster_name[cpid];
		int col = cluster_location[col_cluster];

		for (int rpid = blockIdx.y; rpid < cpid; rpid += gridDim.y) {
			float tmp_x2 = tmp_x - x[rpid];
			float tmp_y2 = tmp_y - y[rpid];
			float tmp_z2 = tmp_z - z[rpid];
			int row_cluster = cluster_name[rpid];
			int row = cluster_location[row_cluster];

			if (row_cluster != col_cluster && norm3df(tmp_x2, tmp_y2, tmp_z2) < threshold) {
				int sub_mat_col = col / BLOCK_SIZE_X;
				int sub_mat_row = row / BLOCK_SIZE_X;

				sample_matrix[sub_mat_col + sub_mat_row * sample_size] = 1;
			}
		}
		__syncthreads();
	}
}

__global__ void buildAdjacencyMatrixM4(float *x, float *y, float *z, int point_num,
										int *cluster_name, int *cluster_location,
										int *sub_mat_location, int sample_size,
										int cluster_num, float threshold, int *matrix)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	int *sub_mat;

	// Loop over columns
	for (int cpid = idx; cpid < point_num; cpid += stride) {
		float tmp_x = x[cpid];
		float tmp_y = y[cpid];
		float tmp_z = z[cpid];
		int col_cluster = cluster_name[cpid];
		int col = cluster_location[col_cluster];

		for (int rpid = blockIdx.y; rpid < cpid; rpid += gridDim.y) {
			float tmp_x2 = tmp_x - x[rpid];
			float tmp_y2 = tmp_y - y[rpid];
			float tmp_z2 = tmp_z - z[rpid];
			int row_cluster = cluster_name[rpid];
			int row = cluster_location[row_cluster];

			if (row_cluster != col_cluster && norm3df(tmp_x2, tmp_y2, tmp_z2) < threshold) {
				// Location of the sub-matrix that contains the cell (row, col)
				int sub_mat_col = col / BLOCK_SIZE_X;
				int sub_mat_row = row / BLOCK_SIZE_X;

				sub_mat = matrix + sub_mat_location[sub_mat_col + sub_mat_row * sample_size] * BLOCK_SIZE_X * BLOCK_SIZE_X;

				// Local row and col of the cell in the sub-matrix
				int local_col = col % BLOCK_SIZE_X;
				int local_row = row % BLOCK_SIZE_X;

				sub_mat[local_col + local_row * BLOCK_SIZE_X] = 1;
			}
		}
	}
}


__global__ void mergeLocalClustersM4(int *cluster_list, int *matrix, int cluster_num, bool *changed,
									int *sample_matrix, int sample_mat_size, int *sub_mat_location)
{
	__shared__ int local_cluster_idx[BLOCK_SIZE_X];
	__shared__ int local_cluster_changed[BLOCK_SIZE_X];
	bool lchanged = false;
	__shared__ bool schanged;

	schanged = false;
	__syncthreads();

	if (blockIdx.x < sample_mat_size && sample_matrix[blockIdx.x + blockIdx.x * sample_mat_size] != 0) {
		// Access the corresponding sub-matrix
		// Column id is also the point id
		int local_col = threadIdx.x;
		int global_col = threadIdx.x + blockIdx.x * BLOCK_SIZE_X;
		int row_end = (blockIdx.x * BLOCK_SIZE_X + BLOCK_SIZE_X <= cluster_num) ? BLOCK_SIZE_X : cluster_num - blockIdx.x * BLOCK_SIZE_X;

		if (global_col < cluster_num) {
			int *sub_mat = matrix + sub_mat_location[blockIdx.x + blockIdx.x * sample_mat_size] * BLOCK_SIZE_X * BLOCK_SIZE_X;

			local_cluster_idx[threadIdx.x] = threadIdx.x;
			__syncthreads();

			for (int local_row = 0; local_row < row_end; local_row++) {
				int col_cluster = local_cluster_idx[local_col];
				int row_cluster = local_cluster_idx[local_row];

				// Reset the 'changed' status of all columns to zero
				local_cluster_changed[local_col] = 0;
				__syncthreads();

				/* If the col and row clusters are different and they are connected,
				 * then the col cluster will be 'marked' to be changed to the row cluster.
				 */
				if (local_row < local_col && col_cluster != row_cluster && sub_mat[local_col + local_row * BLOCK_SIZE_X] == 1) {
					local_cluster_changed[col_cluster] = 1;
					lchanged = true;
				}
				__syncthreads();

				/* If a col cluster X was marked 'changed', change labels of all columns which were labeled
				 * as X to the label of the row cluster
				 */
				local_cluster_idx[local_col] = (local_cluster_changed[col_cluster] == 1) ? row_cluster : col_cluster;
				__syncthreads();
			}

			// Location of the cluster name
			int new_cluster_label = cluster_list[blockIdx.x * BLOCK_SIZE_X + local_cluster_idx[local_col]];
			__syncthreads();

			cluster_list[global_col] = new_cluster_label;

			if (lchanged) {
				schanged = true;
			}
			__syncthreads();

			if (schanged && threadIdx.x == 0) {
				*changed = true;
			}
		}
	}
}


/* Merge clusters that belong to different block of threads*/
__global__ void mergeForeignClustersM4(int *matrix, int *cluster_list,
										int shift_level,
										int sub_mat_size,
										int sub_mat_offset,
										int cluster_num, bool *changed,
										int *sample_matrix, int sample_mat_size, int *sub_mat_location)
{
	// sub_mat_col_base = sub_matrix_size
	// sub_mat_row_base = 0
	int sub_mat_id = blockIdx.x / sub_mat_size;
	int sub_mat_idx = sub_mat_size + sub_mat_id * sub_mat_offset + (shift_level + blockIdx.x) % sub_mat_size;
	int sub_mat_idy = sub_mat_id * sub_mat_offset + blockIdx.x % sub_mat_size;
	bool lchanged = false;
	__shared__ bool schanged;

	__shared__ int cluster_changed[BLOCK_SIZE_X];
	__shared__ int local_clusters[BLOCK_SIZE_X];

	if (threadIdx.x == 0)
		schanged = false;
	__syncthreads();

	if (sub_mat_idx < sample_mat_size && sub_mat_idy < sample_mat_size && sample_matrix[sub_mat_idx + sub_mat_idy * sample_mat_size] == 1) {
		int local_col = threadIdx.x;
		int global_col = local_col + sub_mat_idx * BLOCK_SIZE_X;
		int row_end = (sub_mat_idy * BLOCK_SIZE_X + BLOCK_SIZE_X < cluster_num) ? BLOCK_SIZE_X : cluster_num - sub_mat_idy * BLOCK_SIZE_X;

		if (global_col < cluster_num)
			local_clusters[threadIdx.x] = threadIdx.x;
		__syncthreads();

		int *sub_mat = matrix + sub_mat_location[sub_mat_idx + sub_mat_idy * sample_mat_size] * BLOCK_SIZE_X * BLOCK_SIZE_X;

		for (int local_row = 0; local_row < row_end; local_row++) {
			int col_cluster = local_clusters[threadIdx.x];

			cluster_changed[threadIdx.x] = 0;
			__syncthreads();

			if (sub_mat[local_row * BLOCK_SIZE_X + local_col] == 1) {
				cluster_changed[col_cluster % BLOCK_SIZE_X] = (col_cluster < BLOCK_SIZE_X) ? 1 : 2;
				lchanged = true;
			}
			__syncthreads();

			if ((col_cluster < BLOCK_SIZE_X && cluster_changed[col_cluster] == 1) ||
					(col_cluster >= BLOCK_SIZE_X && cluster_changed[col_cluster - BLOCK_SIZE_X] == 2)) {
				local_clusters[local_col] = local_row + BLOCK_SIZE_X;
			}
			__syncthreads();
		}

		__syncthreads();

		int new_cluster_id = local_clusters[local_col];
		int global_row = threadIdx.x + sub_mat_idy * BLOCK_SIZE_X;

		if (global_row < cluster_num)
			local_clusters[threadIdx.x] = cluster_list[global_row];
		__syncthreads();

		if (new_cluster_id >= BLOCK_SIZE_X) {
			cluster_list[global_col] = local_clusters[new_cluster_id - BLOCK_SIZE_X];
		}

		__syncthreads();

		if (lchanged)
			schanged = true;

		__syncthreads();

		if (threadIdx.x == 0 && schanged)
			*changed = true;
	}
}

__global__ void rebuildSampleMatrixM4(int *old_sample_matrix, int old_size,
									int *old_sub_mat_loc, int *old_matrix,
									int *updated_cluster_list,
									int *new_cluster_location,
									int *new_sample_matrix, int new_sample_size)
{
	for (int i = blockIdx.x; i < old_size * old_size; i += gridDim.x) {
		if (old_sample_matrix[i] > 0) {
			int col = threadIdx.x + (i % old_size) * blockDim.x;
			int new_col = new_cluster_location[updated_cluster_list[col]];

			for (int row = blockIdx.y; row < col; row += gridDim.y) {
				int new_row = new_cluster_location[updated_cluster_list[row]];
				int *old_sub_mat = old_matrix + old_sub_mat_loc[col / BLOCK_SIZE_X + (row / BLOCK_SIZE_X) * old_size] * BLOCK_SIZE_X * BLOCK_SIZE_X;

				if (new_col != new_row && old_sub_mat[(row / BLOCK_SIZE_X) * old_size + (col / BLOCK_SIZE_X)] == 1) {
					new_sample_matrix[(new_col / BLOCK_SIZE_X) + (new_row / BLOCK_SIZE_X) * new_sample_size] = 1;
				}
			}
		}
	}
}

/* Rebuild the adjacency matrix after some clusters are joined together */
__global__ void rebuildAdjacencyMatrixM4(int *old_sample_matrix, int old_size,
										int *old_sub_mat_loc, int *old_matrix,
										int *updated_cluster_list,
										int *new_cluster_location,
										int *new_sub_mat_loc, int new_size, int *new_matrix)
{
	for (int i = blockIdx.x; i < old_size * old_size; i += gridDim.x) {
		if (old_sample_matrix[i] > 0) {
			int col = threadIdx.x + (i % old_size) * blockDim.x;
			int new_col = new_cluster_location[updated_cluster_list[col]];

			for (int row = blockIdx.y; row < col; row += gridDim.y) {
				int new_row = new_cluster_location[updated_cluster_list[row]];
				int *old_sub_mat = old_matrix + old_sub_mat_loc[col / BLOCK_SIZE_X + (row / BLOCK_SIZE_X) * old_size] * BLOCK_SIZE_X * BLOCK_SIZE_X;

				if (new_col != new_row && old_sub_mat[(row / BLOCK_SIZE_X) * old_size + (col / BLOCK_SIZE_X)] == 1) {
					int new_loc = new_sub_mat_loc[(new_row / BLOCK_SIZE_X) * new_size + (new_col / BLOCK_SIZE_X)] * BLOCK_SIZE_X * BLOCK_SIZE_X;

					new_matrix[new_loc + (new_row % BLOCK_SIZE_X) * BLOCK_SIZE_X + (new_col % BLOCK_SIZE_X)] = 1;
				}
			}
		}
	}
}


/* Iterate through the list of remaining clusters and mark the corresponding
 * location on cluster location array by 1
 */
__global__ void clusterMarkM4(int *cluster_list, int *cluster_location, int cluster_num)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = idx; i < cluster_num; i += blockDim.x * gridDim.x) {
		cluster_location[cluster_list[i]] = 1;
	}
}

/* Collect the remaining clusters */
__global__ void clusterCollectorM4(int *old_cluster_list, int *new_cluster_list, int *cluster_location, int cluster_num)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = idx; i < cluster_num; i += blockDim.x * gridDim.x) {
		new_cluster_list[cluster_location[old_cluster_list[i]]] = old_cluster_list[i];
	}
}


/* Rename the cluster name of each point after some clusters are joined together */
__global__ void applyClusterChangedM4(int *cluster_name, int *cluster_list, int *cluster_location, int point_num)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = idx; i < point_num; i += blockDim.x * gridDim.x) {
		int old_cluster = cluster_name[i];

		cluster_name[i] = cluster_list[cluster_location[old_cluster]];
	}
}


void GpuEuclideanCluster2::extractClusters4()
{
	struct timeval start, end;

	// Initialize names of clusters
	initClusters();

	bool *check;
	bool hcheck = false;

	checkCudaErrors(hipMalloc(&check, sizeof(bool)));
	checkCudaErrors(hipMemcpy(check, &hcheck, sizeof(bool), hipMemcpyHostToDevice));

	int block_x, grid_x;

	block_x = (point_num_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : point_num_;
	grid_x = (point_num_ - 1) / block_x + 1;

	gettimeofday(&start, NULL);
	// Divide points into blocks of points and clustering points inside each block
	blockClusteringM4<<<grid_x, block_x>>>(x_, y_, z_, point_num_, cluster_name_, threshold_);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	gettimeofday(&end, NULL);

	std::cout << "blockClustering = " << timeDiff(start, end) << std::endl;

	// Collect the remaining clusters
	// Locations of clusters in the cluster list
	int *cluster_location;

	gettimeofday(&start, NULL);
	checkCudaErrors(hipMalloc(&cluster_location, sizeof(int) * (point_num_ + 1)));
	checkCudaErrors(hipMemset(cluster_location, 0, sizeof(int) * (point_num_ + 1)));
	clusterMarkM4<<<grid_x, block_x>>>(cluster_name_, cluster_location, point_num_);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	int current_cluster_num = 0;
	exclusiveScan(cluster_location, point_num_ + 1, &current_cluster_num);

	int *cluster_list;

	checkCudaErrors(hipMalloc(&cluster_list, sizeof(int) * current_cluster_num));

	clusterCollectorM4<<<grid_x, block_x>>>(cluster_name_, cluster_list, cluster_location, point_num_);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	cluster_num_ = current_cluster_num;

	dim3 grid_size, block_size;

	block_size.x = block_x;
	block_size.y = block_size.z = 1;
	grid_size.x = grid_x;
	grid_size.y = (cluster_num_ > GRID_SIZE_Y) ? GRID_SIZE_Y : cluster_num_;
	grid_size.z = 1;

	// Sample matrix to record the status of each sub-matrix in the big adjacency matrix
	int *sample_matrix;
	int sample_size;

	sample_size = (cluster_num_ - 1) / BLOCK_SIZE_X + 1;
	checkCudaErrors(hipMalloc(&sample_matrix, sizeof(int) * sample_size * sample_size));
	checkCudaErrors(hipMemset(sample_matrix, 0, sizeof(int) * sample_size * sample_size));

	nonZeroSubMatrixCountM4<<<grid_size, block_size>>>(x_, y_, z_, point_num_,
														cluster_name_, cluster_location,
														sample_matrix, sample_size,
														cluster_num_, threshold_);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// Prefix Sum on sample matrix to produce sub-matrix locations
	int *sub_mat_loc;

	checkCudaErrors(hipMalloc(&sub_mat_loc, sizeof(int) * (sample_size * sample_size + 1)));
	checkCudaErrors(hipMemcpy(sub_mat_loc, sample_matrix, sizeof(int) * sample_size * sample_size, hipMemcpyDeviceToDevice));

	int sub_mat_num;

	exclusiveScan(sub_mat_loc, sample_size * sample_size + 1, &sub_mat_num);

	int *matrix;

	checkCudaErrors(hipMalloc(&matrix, sizeof(int) * sub_mat_num * BLOCK_SIZE_X * BLOCK_SIZE_X));
	checkCudaErrors(hipMemset(matrix, 0, sizeof(int) * sub_mat_num * BLOCK_SIZE_X * BLOCK_SIZE_X));


	buildAdjacencyMatrixM4<<<grid_size, block_size>>>(x_, y_, z_, point_num_,
													cluster_name_, cluster_location,
													sub_mat_loc, sample_size,
													cluster_num_, threshold_, matrix);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	gettimeofday(&end, NULL);

	std::cout << "Build RC and Matrix = " << timeDiff(start, end) << std::endl;

	gettimeofday(&start, NULL);
	int itr = 0;

	std::cout << "Cluster num = " << cluster_num_ << std::endl;
	int *matrix_test = (int*)malloc(sizeof(int) * sub_mat_num * BLOCK_SIZE_X * BLOCK_SIZE_X);

	checkCudaErrors(hipMemcpy(matrix_test, matrix, sizeof(int) * sub_mat_num * BLOCK_SIZE_X * BLOCK_SIZE_X, hipMemcpyDeviceToHost));

	for (int i = 0; i < sub_mat_num; i++) {
		for (int j = 0; j < BLOCK_SIZE_X; j++) {
			for (int k = 0; k < BLOCK_SIZE_X; k++) {
				if (matrix_test[i * BLOCK_SIZE_X * BLOCK_SIZE_X + j * BLOCK_SIZE_X + k] != 0)
					std::cout << "(" << j << "," << k << ") ";
			}
		}

		std::cout << std::endl << std::endl;
	}

	free(matrix_test);

	do {
		hcheck = false;

		checkCudaErrors(hipMemcpy(check, &hcheck, sizeof(bool), hipMemcpyHostToDevice));

		int block_x2, grid_x2, grid_y2;

		block_x2 = (cluster_num_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : cluster_num_;
		grid_x2 = (cluster_num_ - 1) / block_x2 + 1;

		mergeLocalClustersM4<<<grid_x2, block_x2>>>(cluster_list, matrix, cluster_num_, check,
													sample_matrix, sample_size, sub_mat_loc);

		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		int sub_matrix_size = 1;
		int sub_matrix_offset = 2;

		checkCudaErrors(hipMemcpy(&hcheck, check, sizeof(bool), hipMemcpyDeviceToHost));


		while (!(hcheck) && sub_matrix_size < cluster_num_ && cluster_num_ > BLOCK_SIZE_X) {

			int sub_matrix_num = (cluster_num_ - 1) / sub_matrix_offset + 1;
			block_x2 = BLOCK_SIZE_X;
			grid_x2 = sub_matrix_size * sub_matrix_num;
			grid_y2 = sub_matrix_size;

			block_size.x = block_x2;
			block_size.y = block_size.z = 1;
			grid_size.x = grid_x2;
			grid_size.y = grid_y2;
			grid_size.z = 1;

			for (int shift_level = 0; shift_level < sub_matrix_size && !(hcheck); shift_level++) {
				mergeForeignClustersM4<<<block_x2, grid_x2>>>(matrix, cluster_list,
															shift_level,
															sub_matrix_size,
															sub_matrix_offset,
															cluster_num_, check,
															sample_matrix, sample_size, sub_mat_loc);
				checkCudaErrors(hipGetLastError());
				checkCudaErrors(hipDeviceSynchronize());

				checkCudaErrors(hipMemcpy(&hcheck, check, sizeof(bool), hipMemcpyDeviceToHost));

			}

			sub_matrix_size *= 2;
			sub_matrix_offset *= 2;
		}


		/* If some changes in the cluster list are recorded (some clusters are merged together),
		 * rebuild the matrix, the cluster location, and apply those changes to the cluster_name array
		 */

		if (hcheck) {
			// Apply changes to the cluster_name array
			applyClusterChangedM4<<<grid_x, block_x>>>(cluster_name_, cluster_list, cluster_location, point_num_);
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());

			checkCudaErrors(hipMemset(cluster_location, 0, sizeof(int) * (point_num_ + 1)));

			block_x2 = (cluster_num_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : cluster_num_;
			grid_x2 = (cluster_num_ - 1) / block_x2 + 1;

			// Remake the cluster location
			clusterMarkM4<<<grid_x2, block_x2>>>(cluster_list, cluster_location, cluster_num_);
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());

			int old_cluster_num = cluster_num_;

			exclusiveScan(cluster_location, point_num_ + 1, &cluster_num_);

			int *new_cluster_list;

			checkCudaErrors(hipMalloc(&new_cluster_list, sizeof(int) * cluster_num_));

			clusterCollectorM4<<<grid_x2, block_x2>>>(cluster_list, new_cluster_list, cluster_location, old_cluster_num);
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());

			// Rebuild matrix
			int *new_sample_matrix;
			int new_sample_size = (cluster_num_ - 1) / BLOCK_SIZE_X + 1;

			checkCudaErrors(hipMalloc(&new_sample_matrix, sizeof(int) * new_sample_size * new_sample_size));
			checkCudaErrors(hipMemset(new_sample_matrix, 0, sizeof(int) * new_sample_size * new_sample_size));

			block_size.x = block_x2;
			block_size.y = block_size.z = 1;
			grid_size.x = grid_x2;
			grid_size.y = (old_cluster_num > GRID_SIZE_Y) ? GRID_SIZE_Y : old_cluster_num;
			grid_size.z = 1;

			rebuildSampleMatrixM4<<<grid_size, block_size>>>(sample_matrix, sample_size,
															sub_mat_loc, matrix,
															cluster_list,
															cluster_location,
															new_sample_matrix, new_sample_size);
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());

			int *new_sub_mat_loc;
			int new_sub_mat_num;

			checkCudaErrors(hipMalloc(&new_sub_mat_loc, sizeof(int) * (new_sample_size * new_sample_size + 1)));
			checkCudaErrors(hipMemcpy(new_sub_mat_loc, new_sample_matrix, sizeof(int) * new_sample_size * new_sample_size, hipMemcpyDeviceToDevice));

			exclusiveScan(new_sub_mat_loc, new_sample_size * new_sample_size + 1, &new_sub_mat_num);

			int *new_matrix;

			checkCudaErrors(hipMalloc(&new_matrix, sizeof(int) * new_sub_mat_num * BLOCK_SIZE_X * BLOCK_SIZE_X));
			checkCudaErrors(hipMemset(new_matrix, 0, sizeof(int) * new_sub_mat_num * BLOCK_SIZE_X * BLOCK_SIZE_X));

			rebuildAdjacencyMatrixM4<<<grid_size, block_size>>>(sample_matrix, sample_size,
																sub_mat_loc, matrix,
																cluster_list,
																cluster_location,
																new_sub_mat_loc, new_sample_size, new_matrix);
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());


			checkCudaErrors(hipFree(cluster_list));
			cluster_list = new_cluster_list;

			checkCudaErrors(hipFree(matrix));
			matrix = new_matrix;

			checkCudaErrors(hipFree(sample_matrix));
			sample_matrix = new_sample_matrix;

			checkCudaErrors(hipFree(sub_mat_loc));
			sub_mat_loc = new_sub_mat_loc;
		}

		std::cout << "Cluster num = " << cluster_num_ << std::endl;

		itr++;
	} while (hcheck);


	gettimeofday(&end, NULL);

	std::cout << "Iteration = " << timeDiff(start, end) << " number of iterations = " << itr << std::endl;

	renamingClusters(cluster_name_, cluster_location, point_num_);

	checkCudaErrors(hipMemcpy(cluster_name_host_, cluster_name_, point_num_ * sizeof(int), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(matrix));
	checkCudaErrors(hipFree(cluster_list));
	checkCudaErrors(hipFree(cluster_location));
	checkCudaErrors(hipFree(check));
	checkCudaErrors(hipFree(sample_matrix));
	checkCudaErrors(hipFree(sub_mat_loc));
}
